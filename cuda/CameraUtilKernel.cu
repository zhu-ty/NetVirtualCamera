#include "hip/hip_runtime.h"
/**
@brief cuda source file of camera utility class
@author: Shane Yuan
@date: Sep 1, 2017
*/

#include "CameraUtilKernel.h"

/**
@brief cuda demosaicing kernel function
@param cv::cuda::PtrStep<uchar> bayerImg: input bayerImg
@param cv::cuda::PtrStep<uchar3> bgrImg: output color image
@param int width: image width
@param int height: image height
*/
__global__ void CameraUtilKernel::demosaic(cv::cuda::PtrStep<uchar> bayerImg,
	cv::cuda::PtrStep<uchar3> bgrImg,
	int width, int height) {
	// get thread position
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	//
	if (x < width / 2 && y < height / 2) {
		int index_row1 = 2 * y * width + 2 * x;
		int index_row2 = index_row1 + width;

		uchar r = bayerImg.ptr(2 * y)[2 * x];
		uchar g1 = bayerImg.ptr(2 * y)[2 * x + 1];
		uchar g2 = bayerImg.ptr(2 * y + 1)[2 * x];	
		uchar b = bayerImg.ptr(2 * y + 1)[2 * x + 1];

		uchar3 g1_uchar3 = make_uchar3(b, g1, r);
		uchar3 r_uchar3 = make_uchar3(b, g1, r);
		uchar3 b_uchar3 = make_uchar3(b, g2, r);
		uchar3 g2_uchar3 = make_uchar3(b, g2, r);

		bgrImg.ptr(2 * y)[2 * x] = g1_uchar3;
		bgrImg.ptr(2 * y)[2 * x + 1] = r_uchar3;
		bgrImg.ptr(2 * y + 1)[2 * x] = b_uchar3;
		bgrImg.ptr(2 * y + 1)[2 * x + 1] = g2_uchar3;
	}
}


/**
@brief demosaic function
@param cv::Mat bayerImg: input bayer image
@return cv::Mat bgrImg: demosaic result
*/
cv::Mat CameraUtilKernel::demosaic(cv::Mat bayerImg) {
	cv::cuda::GpuMat bayerImg_d;
	bayerImg_d.upload(bayerImg);
	cv::cuda::GpuMat gpuImg(bayerImg.rows, bayerImg.cols, CV_8UC3);
	dim3 dimBlock(32, 32);
	dim3 dimGrid((bayerImg.cols / 2 + dimBlock.x - 1) / dimBlock.x,
		(bayerImg.rows / 2 + dimBlock.y - 1) / dimBlock.y);
	CameraUtilKernel::demosaic << <dimGrid, dimBlock >> >(bayerImg_d, gpuImg, bayerImg.cols, bayerImg.rows);
	cv::Mat bgrImg;
	gpuImg.download(bgrImg);
	return bgrImg;
}


